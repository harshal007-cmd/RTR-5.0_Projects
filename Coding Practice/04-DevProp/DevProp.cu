#include<stdio.h>
void PrintCUDADevicePropperties();


int main()
{

	PrintCUDADevicePropperties();

}

void PrintCUDADevicePropperties()
{
	printf("CUDA Info:\n");
	printf("=============================================\n");
	
	hipError_t ret_cuda_rt;
	int dev_count;
	ret_cuda_rt = hipGetDeviceCount(&dev_count);
	if (ret_cuda_rt != hipSuccess)
	{
		printf("CUDA Runtime API Error - hipGetDeviceCount() Failed Due to %s\n", hipGetErrorString(ret_cuda_rt));
	}
	else if (dev_count == 0)
	{
		printf("There is no CUDA Support device on this system\n");
	}
	else
	{
		printf("Total Number of CUDA supporting GPU device/devices on this system : %d\n", dev_count);
		for (int i = 0;i < dev_count;++i)
		{
			hipDeviceProp_t dev_prop;
			int driverVersion = 0, runtimeVersion = 0;

			ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
			if (ret_cuda_rt != hipSuccess)
			{
				printf("%s in %s line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
				return;
			}
			printf("\n");
			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);
			printf("******** CUDA DRIVER AND RUNTIME INFORMATION ********\n");
			printf("=====================================================\n");
			printf("CUDA driver verison                               : %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
			printf("CUDA runtime version                              : %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
			printf("\n");
			printf("=====================================================\n");
			printf("******** GPU DEVICE GENERAL INFORMATION ********\n");
			printf("=====================================================\n");
			printf("GPU Device Number                                 : %d\n", i);
			printf("GPU Device Name                                   : %s\n", dev_prop.name);
			printf("GPU Device Compute Capability                     : %d.%d\n", dev_prop.major, dev_prop.minor);
			printf("GPU Device Clock Rate                             : %d\n", dev_prop.clockRate);
			printf("GPU Device Type                                   : ");
			if (dev_prop.integrated)
				printf("Intergrated (On-Board)\n");
			else
				printf("Discrete (Card)\n");
			
			printf("\n");
			printf("******** GPU DEVICE MEMORY INFORMATION ********\n");
			printf("===============================================\n");
			printf("GPU Device Total Memory                        : %.0f GB = %.0f MB ==%llu bytes\n", ((float)dev_prop.totalGlobalMem / 1048567.0f) / 1024.0f, (float)dev_prop.totalGlobalMem / 1048567.0f, (unsigned long long)dev_prop.totalGlobalMem);
			printf("GPU Device Constat Memory                      : %lu Bytes\n", (unsigned long)dev_prop.totalConstMem);
			printf("GPU Device Shared Memory per SMprocessor       : %lu\n", (unsigned long)dev_prop.sharedMemPerBlock);
			
			printf("\n");
			printf("******** GPU DEVICE MULTIPROCESSOR INFORMATION ********\n");
			printf("=======================================================\n");
			
			printf("GPU Device Number of SMProcessors                : %d\n", dev_prop.multiProcessorCount);
			printf("GPU Device Number of Registers per SMProcessors  : %d\n", dev_prop.regsPerBlock);

			printf("\n");
			printf("******** GPU DEVICE THREAD INFORMATION ********\n");
			printf("===============================================\n");

			printf("GPU Device Maximum Number of Threads Per SMProcessor   : %d\n", dev_prop.maxThreadsPerMultiProcessor);
			printf("GPU Device Maxium Number of Thread per Block           : %d\n", dev_prop.maxThreadsPerBlock);
			printf("GPU device threads in Warp                             : %d\n", dev_prop.warpSize);
			printf("GPU Device Max Thread Dimensions                       : ( %d, %d, %d )\n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
			printf("GPU Device Max Grip Dimensions                         : ( %d, %d, %d )\n", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);

			printf("\n");
			printf("******** GPU DEVIEC DRIVER INFORMATION ********\n");
			printf("===============================================\n");
			printf("GPU Device has ECC support                     : %s\n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");
			
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64) 
			printf("GPU device CUDA Driver Mode (TCC Or WDDM)       : %s\n", dev_prop.tccDriver ? "TCC ( Tesla Computer Cluster Driver )" : "WDDM (Windows Display Driver Model )");
#endif
			printf("***************************************************************************\n");
		}
	}
}













