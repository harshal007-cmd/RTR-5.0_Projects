#include "hip/hip_runtime.h"
#include<stdio.h>
#define CL_TARGET_OPENCL_VERSION 120
#include<hip/hip_runtime.h>
#include"helper_timer.h"

#define BLOCK_WIDTH 32
//global ver


int* hostA = NULL;
int* hostB = NULL;
int* hostC = NULL;
int* gold = NULL;

int* deviceA = NULL;
int* deviceB = NULL;
int* deviceC = NULL;

float timeOnGPU = 0.0f;
float timeOnCPU = 0.0f;

//CUDA Kernel
__global__ void matMulGPU(int* A, int* B, int* C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
	//code
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < numARows) && (column < numBColumns))
	{
		int value = 0.0;
		for (int k = 0;k < numAColumns;k++)
		{
			int a = A[row * numAColumns + column];
			int b = B[k * numBColumns + column];
			value += a * b;
		}
		C[row * numCColumns + column] = value;
	}

}

int main()
{
	void InitA(int* data, int, int);
	void InitB(int* data, int, int);
	void matMulCPU(int*, int*, int*, int, int, int, int);
	void cleanup(void);

	//var declare
	int numARows = BLOCK_WIDTH;
	int numAColumns = BLOCK_WIDTH;
	int numBRows = BLOCK_WIDTH;
	int numBColumns = BLOCK_WIDTH;
	int numCRows = numARows;
	int numCColumns = numBColumns;
	
	int numGoldRows = numARows;
	int numGoldColumns = numBColumns;
	
	int sizeA = numARows * numAColumns * sizeof(int);
	int sizeB = numBRows * numBColumns * sizeof(int);
	int sizeC = numCRows * numCColumns * sizeof(int);
	int sizeGold = numGoldRows * numGoldColumns * sizeof(int);

	hipError_t result = hipSuccess;

	//host memo allocation
	hostA = (int*)malloc(sizeA);
	if (hostA == NULL)
	{
		printf("Host memo allocation failed for host1 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostB = (int*)malloc(sizeB);
	if (hostB == NULL)
	{
		printf("Host memo allocation failed for host2 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostC = (int*)malloc(sizeC);
	if (hostC == NULL)
	{
		printf("Host memo allocation failed for hostoutput array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (int*)malloc(sizeGold);
	if (gold == NULL)
	{
		printf("Host memo allocation failed for gold array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//printing matrix dimentions and sizes
	printf("The Dimensions of Matrix 'hostA' are : %d x %d\n", numARows, numAColumns);
	printf("The Dimensions of Matrix 'hostB' are : %d x %d\n", numBRows, numBColumns);
	printf("The Dimensions of Matrix 'hostC' are : %d x %d\n", numCRows, numCColumns);
	printf("The Dimensions of Matrix 'hostGold' are : %d x %d\n", numGoldRows, numGoldColumns);

	printf("Size of Matrix hostA = %d\n", sizeA);
	printf("Size of Matrix hostB = %d\n", sizeB);
	printf("Size of Matrix hostC = %d\n", sizeC);
	printf("Size of Matrix hostGold = %d\n", sizeGold);

	//fill source matrices
	InitA(hostA, numARows, numAColumns);
	InitB(hostB, numBRows, numBColumns);

	//device memo allocation
	result = hipMalloc((void**)&deviceA, sizeA);
	if (result != hipSuccess)
	{
		printf("Device Memo allocation is failed for deviceA array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	result = hipMalloc((void**)&deviceB, sizeB);
	if (result != hipSuccess)
	{
		printf("Device Memo allocation is failed for deviceB array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void**)&deviceC, sizeC);
	if (result != hipSuccess)
	{
		printf("Device Memo allocation is failed for deviceC array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//copy data from host array to device arrays
	result = hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to Device data copy is failed for deviceA array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}


	result = hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);
	if (result  != hipSuccess)
	{
		printf("Host to Device data copy is failed for deviceB array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}


	//CUDA kernel conf
	dim3 dimGrid = dim3(ceil((int)numBColumns / (int)BLOCK_WIDTH), ceil((int)numARows / (int)BLOCK_WIDTH), 1);
	dim3 dimBlock = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	
	//start timer
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	matMulGPU <<<dimGrid, dimBlock >>> (deviceA, deviceB, deviceC, numARows, numAColumns, numBColumns, numCColumns);

	//stop timer
	sdkStopTimer(&timer);
	timeOnGPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;

	//copy data from device to host
	result = hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{
		printf("Device to host data copy failed for hostC matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//matMul on host
	matMulCPU(hostA, hostB, gold, numARows, numAColumns, numBColumns, numCColumns);


	//comparison
	int breakValue = -1;
	bool bAccuracy = true;
	
	for (int i = 0; i < numCRows*numCColumns; ++i)
	{
		float val1 = gold[i];
		float val2 = hostC[i];
		if (val1 != val2)
		{
			bAccuracy = false;
			breakValue = i;
			break;
		}
	}

	char str[128];
	if (bAccuracy == false)
		sprintf(str, "Comparison of CPU and GPU Matrix Multiplication is not accurate at array index %d", breakValue);
	else
		sprintf(str, "Comparison of CPU and GPU Matrix Multiplication is accurate\n");

	
	//output
	printf("Time taken for Matrix Multiplication on CPU = %.6f\n", timeOnCPU);
	printf("Time taken for Matrix Multiplication on GPU = %.6f\n", timeOnGPU);
	printf("%s\n", str);

	cleanup();

	return 0;
}

void InitA(int* data, int row, int col)
{
	int num = 1;
	for (int i = 0;i < row;i++)
	{
		for (int j = 0;j < col; j++)
		{
			*(data + i * col + j) = num;
			num++;
		}
	}
}

void InitB(int* data, int row, int col)
{
	int num = BLOCK_WIDTH;
	for (int i = 0; i < row; i++)
	{
		for (int j = 0;j < col; j++)
		{
			*(data + i * col + j) = num;
			num--;
		}
	}
}


void matMulCPU(int* A, int* B, int* C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for (int i = 0;i < numARows;++i)
	{
		int value = 0.0f;
		for (int j = 0; i < numBColumns; ++j)
		{
			for (int k = 0;i < numAColumns; ++k)
			{
				int a = A[i * numAColumns + k];
				int b = B[k * numBColumns + j];
				value += a * b;
			}
			C[i * numCColumns + j] = value;
		}
	}

	sdkStopTimer(&timer);
	timeOnCPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;

}

void cleanup()
{
	if (deviceC)
	{
		hipFree(deviceC);
		deviceC = NULL;
	}

	if (deviceB)
	{
		hipFree(deviceB);
		deviceB = NULL;
	}

	if (deviceA)
	{
		hipFree(deviceA);
		deviceA = NULL;
	}

	
	if (hostC)
	{
		free(hostC);
		hostC = NULL;
	}

	if (hostB)
	{
		free(hostB);
		hostB = NULL;
	}

	if (hostA)
	{
		free(hostA);
		hostA = NULL;
	}

}

//cl.exe filename.cpp /I "C:/NVDIA toolkit/include" /Link /LIBPATH: "LIB/x64.../lib" OpenCL.lib /OUT:"filename.exe"












