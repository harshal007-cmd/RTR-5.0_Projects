#include "hip/hip_runtime.h"
#include<stdio.h>
#define CL_TARGET_OPENCL_VERSION 120
#include<hip/hip_runtime.h>
#include"helper_timer.h"

//global ver

const int iNumberOfArrayElements = 11444777;


float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostOutput = NULL;
float* gold = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceOutput = NULL;

float timeOnGPU = 0.0f;
float timeOnCPU = 0.0f;

//CUDA Kernel
__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)
{
	//code
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

int main()
{
	void fillFloatArrayWithRandomNumbers(float*, int);
	void vecAddCPU(const float*, const float*, float*, int);
	void cleanup(void);

	//var declare
	int size = iNumberOfArrayElements * sizeof(float);
	hipError_t result = hipSuccess;

	//host memo allocation
	hostInput1 = (float*)malloc(size);
	if (hostInput1 == NULL)
	{
		printf("Host memo allocation failed for host1 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float*)malloc(size);
	if (hostInput2 == NULL)
	{
		printf("Host memo allocation failed for host2 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostOutput = (float*)malloc(size);
	if (hostOutput == NULL)
	{
		printf("Host memo allocation failed for hostoutput array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (float*)malloc(size);
	if (gold == NULL)
	{
		printf("Host memo allocation failed for gold array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//filling values in host array
	fillFloatArrayWithRandomNumbers(hostInput1, iNumberOfArrayElements);
	fillFloatArrayWithRandomNumbers(hostInput2, iNumberOfArrayElements);

	//device memo allocation
	result = hipMalloc((void**)&deviceInput1, size);
	if (result != hipSuccess)
	{
		printf("Device Memo allocation is failed for deviceInput1 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	result = hipMalloc((void**)&deviceInput2, size);
	if (result != hipSuccess)
	{
		printf("Device Memo allocation is failed for deviceInput2 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void**)&deviceOutput, size);
	if (result != hipSuccess)
	{
		printf("Device Memo allocation is failed for deviceOutput array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//copy data from host array to device arrays
	result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to Device data copy is failed for deviceInput1 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}


	result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if (result  != hipSuccess)
	{
		printf("Host to Device data copy is failed for deviceInput2 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}


	//CUDA kernel conf
	dim3 dimGrid = dim3((int)ceil((float)iNumberOfArrayElements / 1));
	dim3 dimBlock = dim3(256, 1, 1);

	
	//start timer
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	vecAddGPU << <dimGrid, dimBlock >> > (deviceInput1, deviceInput2, deviceOutput, iNumberOfArrayElements);

	//stop timer
	sdkStopTimer(&timer);
	timeOnGPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;



	//vector addition on host
	vecAddCPU(hostInput1, hostInput2, gold, iNumberOfArrayElements);

	//comparison
	const float epsilon = 0.000001f;
	int breakValue = -1;
	bool bAccuracy = true;
	
	for (int i = 0;i < iNumberOfArrayElements;++i)
	{
		float val1 = gold[i];
		float val2 = hostOutput[i];
		if (fabs(val1 - val2) > epsilon)
		{
			bAccuracy = false;
			breakValue = i;
			break;
		}
	}

	char str[128];
	if (bAccuracy == false)
		sprintf(str, "Comparison of CPU and GPU vector addition is not within accuracy of 0.000001 at array index %d", breakValue);
	else
		sprintf(str, "Comparison of CPu and GPU vector addtion is within accuracy of 0.000001");

	
	//output
	printf("Array1 begins from 0th index %.6f to %dth index %.6f\n", hostInput1[0], iNumberOfArrayElements - 1, hostInput1[iNumberOfArrayElements - 1]);
	printf("Array1 begins from 0th index %.6f to %dth index %.6f\n", hostInput2[0], iNumberOfArrayElements - 1, hostInput2[iNumberOfArrayElements - 1]);
	
	printf("Output array begins from 0th index %.6f to %dth index %.6f\n", hostOutput[0], iNumberOfArrayElements - 1, hostOutput[iNumberOfArrayElements - 1]);

	printf("Time taken for Vector additional on CPU = %.6f\n", timeOnCPU);
	printf("Time taken for Vector additional on GPU = %.6f\n", timeOnGPU);
	printf("%s\n", str);

	cleanup();

	return 0;
}

void fillFloatArrayWithRandomNumbers(float* arr, int len)
{
	const float fscale = 1.0f / (float)RAND_MAX;
	for (int i = 0;i < len;i++)
	{
		arr[i] = fscale * rand();
	}

}

void vecAddCPU(const float* arr1, const float* arr2, float* out, int len)
{
	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for (int i = 0;i < len;++i)
	{
		out[i] = arr1[i] + arr2[i];
	}

	sdkStopTimer(&timer);
	timeOnCPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;

}

size_t roundGlobalSizeToNearestMultipleOfLocalSize(int local_size, unsigned int global_size)
{
	unsigned int r = global_size % local_size;
	if (r == 0)
	{
		return global_size;
	}
	else
	{
		return (global_size + local_size - r);
	}

}

void cleanup()
{
	if (deviceOutput)
	{
		hipFree(deviceOutput);
		deviceOutput = NULL;
	}

	if (deviceInput2)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}

	if (deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}

	
	if (hostOutput)
	{
		free(hostOutput);
		hostOutput = NULL;
	}

	if (hostInput2)
	{
		free(hostInput2);
		hostInput2 = NULL;
	}

	if (hostInput1)
	{
		free(hostInput1);
		hostInput1 = NULL;
	}

}

//cl.exe filename.cpp /I "C:/NVDIA toolkit/include" /Link /LIBPATH: "LIB/x64.../lib" OpenCL.lib /OUT:"filename.exe"












