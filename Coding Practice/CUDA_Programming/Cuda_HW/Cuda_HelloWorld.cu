#include<stdio.h>
#include<hip/hip_runtime.h>

int main()
{
	hipDeviceProp_t pCuda;
	int cnt;

	hipGetDeviceCount(&cnt);

	for(int i;i < cnt; ++i)
	{
		hipGetDeviceProperties(&pCuda, i);
		printf("Name = %s\n",pCuda.name);
		printf("Compute Capability = %d.%d\n", pCuda.major, pCude.minor);
	}
	return 0;
}
