#include<stdio.h>
#include<hip/hip_runtime.h>

int main()
{
	hipDeviceProp_t pCuda;
	int cnt;

	hipGetDeviceCount(&cnt);

	for(int i=0;i < cnt; ++i)
	{
		hipGetDeviceProperties(&pCuda, i);
		printf("Name = %s\n",pCuda.name);
		printf("Compute Capability = %d.%d\n", pCuda.major, pCuda.minor);
	}
	return 0;
}
