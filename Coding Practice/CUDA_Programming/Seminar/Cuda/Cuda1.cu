#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

const int iNumElement = 5;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostInput3 = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceInput = NULL;

__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

int main(void)
