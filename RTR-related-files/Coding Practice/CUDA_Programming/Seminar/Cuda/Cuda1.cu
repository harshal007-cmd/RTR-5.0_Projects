#include<stdio.h>
#include<hip/hip_runtime.h>

const int iNumElement = 5;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostInput  = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceInput = NULL;

__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

int main(void)
{
	void cleanup(void);

	int size = iNumElement * sizeof(float);
	hipError_t result = hipSuccess;

	hostInput1 = (float*)malloc(size);
	if (hostInput1 == NULL)
	{
		printf("Host memory allocation is falied for hostInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float*)malloc(size);
	if (hostInput2 == NULL)
	{
		printf("Host memory allocation is falied for hostInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput = (float*)malloc(size);
	if (hostInput == NULL)
	{
		printf("Host memory allocation is falied for hostInput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//inputs
	hostInput1[0] = 101.0;
	hostInput1[1] = 102.0;
	hostInput1[2] = 103.0;
	hostInput1[3] = 104.0;
	hostInput1[4] = 105.0;

	hostInput2[0] = 201.0;
	hostInput2[1] = 202.0;
	hostInput2[2] = 203.0;
	hostInput2[3] = 204.0;
	hostInput2[4] = 205.0;

	result = hipMalloc((void**)&deviceInput1, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation failed for dev1\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void**)&deviceInput2, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation failed for dev2\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	result = hipMalloc((void**)&deviceInput, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation failed for dev\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//copy data from host array to device array
	result = hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to device data is failed for dev1 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to device data is failed for dev2 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	dim3 dimGrid = dim3(iNumElement, 1, 1);
	dim3 dimBlock = dim3(1, 1, 1);

	//CUDA karnel for vector addition
	vecAddGPU <<< dimGrid,dimBlock >>> (deviceInput1, deviceInput2, deviceInput, iNumElement);
	
	//copy data to array
	result = hipMemcpy(hostInput,deviceInput,size,hipMemcpyHostToDevice);
	
	if(result != hipSuccess)
	{
		printf("Device to host copy is failed \n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	for(int i=0; i < iNumElement;++i)
	{
		printf("%f + %f = %f\n",hostInput1[i],hostInput2[i],hostInput[i]);
	}

	cleanup();
	return 0;


}

void cleanup()
{
	if(deviceInput)
	{
		hipFree(deviceInput);
		deviceInput = NULL;
	}
	
	if(deviceInput)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}
	if(deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}
	
	if(hostInput)
	{
		hipFree(hostInput);
		hostInput=NULL;
	}
	
	if(hostInput2)
	{
		hipFree(hostInput2);
		hostInput2=NULL;
	}
	if(hostInput1)
	{
		hipFree(hostInput1);
		hostInput1 = NULL;
	}
}








