#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include"helper_timer.h"

const int iNumElement = 5;

float* hostInput1 = NULL;
float* hostInput2 = NULL;
float* hostInput  = NULL;
float* gold = NULL;

float* deviceInput1 = NULL;
float* deviceInput2 = NULL;
float* deviceInput = NULL;

float timeOnCPU = 0.0;
float timeOnGPU = 0.0;

__global__ void vecAddGPU(float* in1, float* in2, float* out, int len)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

int main(void)
{
	void cleanup(void);
	void fillFloatArrayWithRandomNumber(float*, int);
	void vecAddCPU(const float*, const float*, float*, int);

	int size = iNumElement * sizeof(flaot);
	hipError_t result_t = hipSuccess;

	hostInput1 = (float*)malloc(size);
	if (hostInput1 = NULL)
	{
		printf("Host memory allocation is falied for hostInput1 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput2 = (float*)malloc(size);
	if (hostInput2 = NULL)
	{
		printf("Host memory allocation is falied for hostInput2 array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	hostInput = (float*)malloc(size);
	if (hostInput = NULL)
	{
		printf("Host memory allocation is falied for hostInput array.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//inputs
	hostInput1[0] = 101.0;
	hostInput1[1] = 102.0;
	hostInput1[2] = 103.0;
	hostInput1[3] = 104.0;
	hostInput1[4] = 105.0;

	hostInput2[0] = 201.0;
	hostInput2[1] = 202.0;
	hostInput2[2] = 203.0;
	hostInput2[3] = 204.0;
	hostInput2[4] = 205.0;

	result = hipMalloc((void**)&deviceInput1, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation failed for dev1\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void**)&deviceInput2, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation failed for dev2\n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	result = hipMalloc((void**)&deviceInput, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation failed for dev\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//copy data from host array to device array
	result hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to device data is failed for dev1 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to device data is failed for dev2 array\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	dim3 dimGrid = dim3((int)ceil((float)iNumElement/256.0), 1, 1);
	dim3 dimBlock = dim3(256, 1, 1);

	StopWatchInterface* timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);


	//CUDA karnel for vector addition
	vecAddGPU <<< dimGrid,dimBlock >>> (deviceInput1, deviceInput2, deviceInput, iNumElement);
	
	sdkStopTimer(&timer);
	timeOnGPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;
	
	//copy data to array
	result = hipMemcpy(hostInput,deviceInput,size,hipMemcpyHostToDevice);
	
	if(result != hipSuccess)
	{
		printf("Device to host copy is failed \n");
		cleanup();
		exit(EXIT_FAILURE);
	}
	
	vecAddCPU(hostInput1, hostInput2, gold, iNumElement);

	//comparison
	const float epsilon = 0.000001f;
	int breakValue = -1;
	bool bAccuracy = true;
	for (int i = 0; i < iNumElement; ++i)
	{
		float val1 = gold[i];
		float val2 = hostInput[i];
		if (fabs(val1 - val2) > epsilon)
		{
			bAccuracy = false;
			breakValue = i;
			break;
		}
	}

	for(int i=0,i<iNumElement;++i)
	{
		printf("%f + %f = %f\n",hostInput1[i],hostInput2[i],hostInput[i]);
	}

	cleanup();
	return 0;


}

void cleanup()
{
	if(deviceInput)
	{
		hipFree(deviceInput);
		deviceInput = NULL;
	}
	
	if(deviceInput)
	{
		hipFree(deviceInput2);
		deviceInput2 = NULL;
	}
	if(deviceInput1)
	{
		hipFree(deviceInput1);
		deviceInput1 = NULL;
	}
	
	if(hostInput)
	{
		hipFree(hostInput);
		hostInput=NULL;
	}
	
	if(hostInput2)
	{
		hipFree(hostInput2);
		hostInput2=NULL;
	}
	if(hostInput1)
	{
		hipFree(hostInput1);
		hostInput1 = NULL;
	}
}








